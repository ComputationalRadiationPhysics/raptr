/** @file test_cusparseWrapper.cu
 * Author: malte
 *
 * Created on 3. Februar 2015, 13:55
 */

#include <cstdlib>
#include <hipsparse.h>

#include "CUDA_HandleError.hpp"
#include "CUSPARSE_HandleError.hpp"
#include "cusparseWrapper.hpp"

#define NNZ 9
#define M 4



int main(int argc, char** argv) {
  /* Create sparse COO matrix on host
   *   1      
   * 5   4   6
   *   3   2  
   * 8     7 9
   */
  float cooVal_host[NNZ] = {1, 5, 4, 6, 3, 2, 8, 7, 9};
  int cooRowId_host[NNZ] = {0, 1, 1, 1, 2, 2, 3, 3, 3};
  int cooColId_host[NNZ] = {1, 0, 2, 4, 1, 3, 0, 3, 4};
  
  /* Copy COO matrix to device */
  float * cooVal_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&cooVal_devi, sizeof(cooVal_devi[0]) * NNZ));
  HANDLE_ERROR(hipMemcpy(cooVal_devi, cooVal_host, sizeof(cooVal_devi[0]) * NNZ, hipMemcpyHostToDevice));
  int * cooRowId_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&cooRowId_devi, sizeof(cooRowId_devi[0]) * NNZ));
  HANDLE_ERROR(hipMemcpy(cooRowId_devi, cooRowId_host, sizeof(cooRowId_devi[0]) * NNZ, hipMemcpyHostToDevice));
  int * cooColId_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&cooColId_devi, sizeof(cooColId_devi[0]) * NNZ));
  HANDLE_ERROR(hipMemcpy(cooColId_devi, cooColId_host, sizeof(cooColId_devi[0]) * NNZ, hipMemcpyHostToDevice));
  
  
  hipsparseHandle_t handle = NULL;
  HANDLE_CUSPARSE_ERROR(
        hipsparseCreate(&handle));
  
  hipsparseMatDescr_t descr = NULL;
  HANDLE_CUSPARSE_ERROR(
        hipsparseCreateMatDescr(&descr));
  HANDLE_CUSPARSE_ERROR(
        customizeMatDescr(descr, handle));
  
  int * csrRowPtr_devi = NULL; 
  HANDLE_ERROR(hipMalloc((void**)&csrRowPtr_devi, sizeof(csrRowPtr_devi[0]) * (M+1)));
  HANDLE_CUSPARSE_ERROR(
        convertCoo2Csr(csrRowPtr_devi, cooRowId_devi, handle, NNZ, M));
          
  hipFree(cooVal_devi);
  hipFree(cooRowId_devi);
  hipFree(cooColId_devi);
  hipsparseDestroy(handle);
  hipsparseDestroyMatDescr(descr);
  hipFree(csrRowPtr_devi);
  
  
  return 0;
}

