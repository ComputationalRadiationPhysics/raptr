/**
 * Copyright 2016 Malte Zacharias
 *
 * This file is part of raptr.
 *
 * raptr is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * raptr is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with raptr.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/** @file test_MeasurementSetup.cu */
#include "hip/hip_runtime.h"
#include "MeasurementSetup.hpp"
#include <iostream>
#include <string>
#include "H5Reader.hpp"

#define NA  180
#define N0Z 13
#define N0Y 13
#define N1Z 13
#define N1Y 13
#define NCHANNELS NA*N0Z*N0Y*N1Z*N1Y 

typedef float val_t;

int main(int ac, char ** av)
{
  /* Treat command line args */
  if(ac < 2)
  {
    std::cerr << "too few args" << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string fn(av[1]);
  
  /* Read measurement data */
  H5Reader h5reader(fn);
  val_t * meas = new val_t[NCHANNELS];
  h5reader.read(meas);

  /* Create setup object */
  DefaultMeasurementSetup<val_t>
        setup(-1., 1.,
              NA, N0Z, N0Y, N1Z, N1Y,
              2, 0.1, 0.1, 0.1);

  /* Print data */
  for(int i=0; i<NCHANNELS; i++)
  {
    if(meas[i] != 0.)
    {
      int sepCnlId[5] = {0,0,0,0,0};
      setup.sepChannelId(sepCnlId, i);
      std::cout << "ida: "  << sepCnlId[0] << ", "
                << "id0z: " << sepCnlId[1] << ", "
                << "id0y: " << sepCnlId[2] << ", "
                << "id1z: " << sepCnlId[3] << ", "
                << "id1y: " << sepCnlId[4] << "    "
                << meas[i] << std::endl;
    }
  }
  std::cout << std::endl;

  //int const sepCnlId[] = {1, 0, 0, 0, 0};
  //int const linCnlId(setup.linChannelId(sepCnlId));
  //int sepCnlId_[5];
  //setup.sepChannelId(sepCnlId_, linCnlId);

  //std::cout << linCnlId << std::endl
  //          << sepCnlId_[0] << " "
  //          << sepCnlId_[1] << " "
  //          << sepCnlId_[2] << " "
  //          << sepCnlId_[3] << " "
  //          << sepCnlId_[4] << " "
  //          << std::endl;
  
  return 0;
}
