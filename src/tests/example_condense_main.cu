#include "hip/hip_runtime.h"
/**
 * Copyright 2016 Malte Zacharias
 *
 * This file is part of raptr.
 *
 * raptr is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * raptr is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with raptr.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/** @file example_condense_main.cu */
/* 
 * File:   example_condense_main.cu
 * Author: malte
 *
 * Created on 26. November 2014, 16:15
 */

#include <cstdlib>
#include <vector>
#include <algorithm>
#include <string>
#include <fstream>

#include "CUDA_HandleError.hpp"
#include "example_condense.h"

using namespace std;

/*
 * 
 */
int main(int argc, char** argv) {
  int const nargs(2);
  if(argc!=nargs+1) {
    std::cerr << "Error: Wrong number of arguments. Exspected " << nargs
              << ":" << std::endl
              << "    output mode (b: binary/c: charakter)" << std::endl
              << "    output filename" << std::endl
              ;
    exit(EXIT_FAILURE);
  }
  std::string const mode(argv[1]);
  std::string const out_fn(argv[2]);
  if((mode!=string("b"))&&(mode!=string("c"))) {
    std::cerr << "Error: Invalid mode specification (b/c)" << std::endl;
    exit(EXIT_FAILURE);
  }
  
  std::vector<val_t> passed_host(SIZE, 0.);
  int   memId_host[1] = {0};
  
  val_t * passed_devi = NULL;
  int *   memId_devi  = NULL;
  HANDLE_ERROR(hipMalloc((void**)&passed_devi, sizeof(passed_devi[0]) * SIZE));
  HANDLE_ERROR(hipMalloc((void**)&memId_devi,  sizeof(memId_devi[0])));
  HANDLE_ERROR(hipMemcpy(passed_devi, &passed_host[0], sizeof(passed_devi[0]) * SIZE, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(memId_devi,  &memId_host[0],  sizeof(memId_devi[0]),         hipMemcpyHostToDevice));
  
  condense<<<NBLOCKS, TPB>>>(passed_devi, memId_devi);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  HANDLE_ERROR(hipMemcpy(&passed_host[0], passed_devi, sizeof(passed_host[0]) * SIZE, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(&memId_host[0],  memId_devi,  sizeof(memId_host[0]),         hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipDeviceSynchronize());
  
//  std::cout << "Found: " << *memId_host << std::endl;
//  for(int i=0; i<*memId_host; i++) {
//    std::cout << "passed[" << i 
//              << "]: " << passed_host[i]
//              << ", stuff: " << stuff_host[i]
//              << ", block: " << block_host[i]
//              << std::endl;
//  }
  
  std::sort(passed_host.begin(), passed_host.end());
  std::ofstream out(out_fn.c_str(), std::ofstream::trunc|std::ios_base::binary);
  if(!out.is_open()) {
    std::cerr << "Error: Could not open file " << out_fn << std::endl;
    
    hipFree(passed_devi);
    hipFree(memId_devi);
    
    exit(EXIT_FAILURE);
  }
  
  if(mode==string("b")) {
    for(int i=0; i<passed_host.size(); i++) {
      out.write((char*)&passed_host[i], sizeof(passed_host[0]));
    }
  } else {
    for(int i=0; i<passed_host.size(); i++) {
      out << passed_host[i];
    }
  }
  
  hipFree(passed_devi);
  hipFree(memId_devi);
  
  return 0;
}

