/**
 * Copyright 2016 Malte Zacharias
 *
 * This file is part of raptr.
 *
 * raptr is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * raptr is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with raptr.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/** @file test_getSystemMatrixFromWorkqueue.cu */
/* 
 * Author: malte
 *
 * Created on 22. Oktober 2014, 10:29
 */

#include <cstdlib>
#include "FileTalk.hpp"
#include "getSystemMatrixFromWorkqueue.cu"
#include "getWorkqueue.hpp"
#include "VoxelGrid.hpp"
#include "MeasurementSetup.hpp"
#include "MeasurementSetupLinIndex.hpp"
#include "MeasurementSetupTrafo2CartCoord.hpp"
#include "H5File2DefaultMeasurementList.h"
#include "H5DensityWriter.hpp"
#include "GridAdapter.hpp"
#include "real_measurementsetup_defines.h"
#include "voxelgrid_defines.h"
#include <iostream>
#include "CUDA_HandleError.hpp"

#include "typedefs.hpp"
#include "device_constant_memory.hpp"

/*
 * Simple C++ Test Suite
 */

#define NBLOCKS 32
#define TPB 256



int main(int argc, char** argv) {
  int const nargs(3);
  if(argc!=nargs+1) {
    std::cerr << "Error: Wrong number of arguments. Exspected: "
              << nargs << ":" << std::endl
              << "  filename of measurement" << std::endl
              << "  filename of output" << std::endl
              << "  number of rays" << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string const fn(argv[1]);
  std::string const on(argv[2]);
  int const nrays(atoi(argv[3]));

  MS setup =
    MS(
      POS0X, POS1X,
      NA, N0Z, N0Y, N1Z, N1Y,
      DA, SEGX, SEGY, SEGZ);
  
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(setup_const), &setup, sizeof(MS)));
  
  VG grid =
    VG(
      GRIDOX, GRIDOY, GRIDOZ,
      GRIDDX, GRIDDY, GRIDDZ,
      GRIDNX, GRIDNY, GRIDNZ);
  
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(grid_const), &grid, sizeof(grid)));
  
  ML list =
    H5File2DefaultMeasurementList<val_t>(fn, NA*N0Z*N0Y*N1Z*N1Y);
  
  // Allocate memory for workqueue on host
  SAYLINE(__LINE__-1);
  std::vector<int>   wqCnlId_host;;
  std::vector<int>   wqVxlId_host;
  
  // Get Workqueue
  SAYLINE(__LINE__-1);
  int listId(0); int vxlId(0);
  int nFound =
    getWorkqueue<
          val_t,
          ML,
          VG, Idx, Idy, Idz,
          MS, Id0z, Id0y, Id1z, Id1y, Ida,
          Trafo0, Trafo1> (
          wqCnlId_host, wqVxlId_host, listId, vxlId, &list, &grid, &setup);
  
  // Allocate memory for sparse matrix (=workqueue + matrix values) on device
  int * wqCnlId_devi = NULL;
  int * wqVxlId_devi = NULL;
  val_t *   val_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&wqCnlId_devi, sizeof(wqCnlId_devi[0]) *nFound));
  HANDLE_ERROR(hipMalloc((void**)&wqVxlId_devi, sizeof(wqVxlId_devi[0]) *nFound));
  HANDLE_ERROR(hipMalloc((void**)&val_devi,     sizeof(val_devi[0])     *nFound));
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Copy Workqueue to device
  SAYLINE(__LINE__-1);
  HANDLE_ERROR(hipMemcpy(
        wqCnlId_devi, &(*wqCnlId_host.begin()), sizeof(wqCnlId_devi[0]) *nFound, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(
        wqVxlId_devi, &(*wqVxlId_host.begin()), sizeof(wqVxlId_devi[0]) *nFound, hipMemcpyHostToDevice));
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Kernel launch
  SAYLINE(__LINE__-1);
  getSystemMatrixFromWorkqueue<
        val_t, VG, Idx, Idy, Idz, MS, Id0z, Id0y, Id1z, Id1y, Ida, Trafo0, Trafo1>
        <<<NBLOCKS, TPB>>> (
        wqCnlId_devi, wqVxlId_devi, val_devi, nFound, nrays);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Allocate memory for matrix values on host
  std::vector<val_t> val_host(nFound, 0);
  
  // Copy matrix values to host
  SAYLINE(__LINE__-1);
  HANDLE_ERROR(hipGetLastError());
  HANDLE_ERROR(hipMemcpy(
        &(*val_host.begin()), val_devi, sizeof(val_host[0]) * nFound, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Sum up values
  SAYLINE(__LINE__-1);
  val_t sum(0);
  for(int i=0; i<nFound; i++) {
    sum += val_host[i];
  }
  std::cout << "Sum is: " << sum << std::endl;
  
  
  // Create grid memory for backprojection
  SAYLINE(__LINE__-1);
  int const gridsize(grid.gridnx()*grid.gridny()*grid.gridnz());
  val_t * mem = new val_t[gridsize];
  for(int vxlId=0; vxlId<gridsize; vxlId++) {
    mem[vxlId] = 0.;
  }
  
  // Backproject workqueue on grid
  SAYLINE(__LINE__-1);
  for(int wqId=0; wqId<nFound; wqId++) {
    int vxlId   = wqVxlId_host[wqId];
    mem[vxlId] += val_host[    wqId];
  }
  
  // Write to hdf5
  SAYLINE(__LINE__-1);
  H5DensityWriter<GridAdapter<VG, val_t> > writer(on);
  GridAdapter<VG, val_t> ga(&grid);
  writer.write(mem, ga);
  
  return (EXIT_SUCCESS);
}

