/** @file test_convertCsr2Ecsr.cu */
#include <iostream>
#include "convertCsr2Ecsr.hpp"
#include "CUDA_HandleError.hpp"

int main() {
  int yRid_host[] = {2, 4, 6};
  int aRptr_host[] = {0, 0, 0, 3, 3, 3, 3, 5};
  
  int * yRid_devi = NULL;
  HANDLE_ERROR(
        hipMalloc((void**)&yRid_devi, sizeof(yRid_devi[0]) * 3));
  HANDLE_ERROR(
        hipMemcpy(yRid_devi, yRid_host, sizeof(yRid_devi[0]) * 3, hipMemcpyHostToDevice));
  int * aRptr_devi = NULL;
  HANDLE_ERROR(
        hipMalloc((void**)&aRptr_devi, sizeof(aRptr_devi[0]) * 8));
  HANDLE_ERROR(
        hipMemcpy(aRptr_devi, aRptr_host, sizeof(aRptr_devi[0]) * 8, hipMemcpyHostToDevice));
   
  int aERptr_host[4];
  int * aERptr_devi = NULL;
  HANDLE_ERROR(
        hipMalloc((void**)&aERptr_devi, sizeof(aERptr_devi[0]) * 4));
  
  convertCsr2Ecsr(aERptr_devi, yRid_devi, 3, aRptr_devi, 7);
  
  HANDLE_ERROR(
        hipMemcpy(aERptr_host, aERptr_devi, sizeof(aRptr_devi[0]) * 4, hipMemcpyDeviceToHost));
  
  for(int i=0; i<4; i++) {
    std::cout << aERptr_host[i] << std::endl;
  }
  
  return 0;
}
