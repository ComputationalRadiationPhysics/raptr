#include "hip/hip_runtime.h"
/** @file backprojection.cu */
/* Author: malte
 *
 * Created on 16. Februar 2015, 11:28 */

#define NBLOCKS 32

#include "cuda_wrappers.hpp"
#include "wrappers.hpp"
#include "CUDA_HandleError.hpp"
#include "CUSPARSE_HandleError.hpp"
#include "measure_time.hpp"
#include "typedefs.hpp"
#include "csrmv.hpp"
#include "mlemOperations.hpp"
#include "RayGenerators.hpp"

/* [512 * 1024 * 1024 / 4] (512 MiB of float or int); max # of elems in COO
 * matrix arrays on GPU */
MemArrSizeType const LIMNNZ(134217728);

/* Max # of channels in COO matrix arrays */
ListSizeType const LIMM(LIMNNZ/VGRIDSIZE);

int main(int argc, char** argv) {
#if MEASURE_TIME
  clock_t time1 = clock();
#endif /* MEASURE_TIME */
  int const nargs(3);
  if(argc!=nargs+1) {
    std::cerr << "Error: Wrong number of arguments. Exspected: "
              << nargs << ":" << std::endl
              << "  filename of measurement" << std::endl
              << "  filename of output" << std::endl
              << "  number of rays" << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string const fn(argv[1]);
  std::string const on(argv[2]);
  
  /* NUMBER OF RAYS PER CHANNEL */
  int const nrays(atoi(argv[3]));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nrays_const), &nrays, sizeof(int)));
  
  /* MEASUREMENT SETUP */
  MS setup = MS(POS0X, POS1X, NA, N0Z, N0Y, N1Z, N1Y, DA, SEGX, SEGY, SEGZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(setup_const), &setup, sizeof(MS)));
  
  /* VOXEL GRID */
  VG grid = VG(GRIDOX, GRIDOY, GRIDOZ, GRIDDX, GRIDDY, GRIDDZ, GRIDNX, GRIDNY, GRIDNZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(grid_const), &grid, sizeof(grid)));
  
  /* MEASUREMENT VECTOR Y */
  /* Number of non-zeros, row indices, values. */
  ListSizeType effM; std::vector<int> yRowId_host; std::vector<val_t> yVal_host;
  
  {
    int tmp_effM(0);
    readHDF5_MeasVct(yRowId_host, yVal_host, tmp_effM, fn);
    effM = ListSizeType(tmp_effM);
  };
  
  int * yRowId_devi = NULL;
  val_t * yVal_devi = NULL;
  mallocD_SparseVct(yRowId_devi, yVal_devi, effM);
  cpyH2DAsync_SparseVct(yRowId_devi, yVal_devi, &yRowId_host[0], &yVal_host[0], effM);

  
  /* STUFF FOR MV */
  hipsparseHandle_t handle = NULL; hipsparseMatDescr_t A = NULL;
  HANDLE_CUSPARSE_ERROR(hipsparseCreate(&handle));
  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&A));
  HANDLE_CUSPARSE_ERROR(customizeMatDescr(A, handle));
  val_t alpha = 1.; val_t beta = 1.;
  
  /* MAX NUMBER OF NON_ZEROS IN SYSTEM MATRIX */
  MemArrSizeType maxNnz(MemArrSizeType(effM) * MemArrSizeType(VGRIDSIZE));
    
  /* DENSITY X */
  std::vector<val_t> x_host(VGRIDSIZE, 0.);
  val_t * x_devi = NULL;
  mallocD<val_t>(x_devi, VGRIDSIZE);
  memcpyH2D<val_t>(x_devi, &x_host[0], VGRIDSIZE);
  
  /* SYSTEM MATRIX */
  /* Row (channel) ids, row pointers, effective row pointers, column (voxel)
   * ids, values, number of non-zeros (host, devi) */
  int * aCnlId_devi = NULL; int * aCsrCnlPtr_devi = NULL;
  int * aEcsrCnlPtr_devi = NULL; int * aVxlId_devi = NULL;
  val_t * aVal_devi = NULL;
  mallocD_SystemMatrix<val_t>(aCnlId_devi, aCsrCnlPtr_devi,
        aEcsrCnlPtr_devi, aVxlId_devi, aVal_devi, NCHANNELS, LIMM, VGRIDSIZE);
  MemArrSizeType * nnz_devi = NULL;
  mallocD<MemArrSizeType>(nnz_devi,          1);
#if MEASURE_TIME
  clock_t time2 = clock();
  printTimeDiff(time2, time1, "Time before BP: ");
#endif /* MEASURE_TIME */
  
  /* BACKPROJECT */
  ChunkGridSizeType NChunks(nChunks<ChunkGridSizeType, MemArrSizeType>
        (maxNnz, MemArrSizeType(LIMM)*MemArrSizeType(VGRIDSIZE))
  );
  for(ChunkGridSizeType chunkId=0;
        chunkId<NChunks;
        chunkId++) {
    ListSizeType m   = nInChunk(chunkId, effM, LIMM);
    ListSizeType ptr = chunkPtr(chunkId, LIMM);
    
    MemArrSizeType nnz_host[1] = {0};
    memcpyH2D<MemArrSizeType>(nnz_devi, nnz_host, 1);
    
    /* Get system matrix */
    systemMatrixCalculation<val_t, ListSizeType, int, MemArrSizeType,
            RandRayGen<val_t, Trafo0_inplace, Trafo1_inplace> > (
          aEcsrCnlPtr_devi, aVxlId_devi, aVal_devi,
          nnz_devi,
          aCnlId_devi, aCsrCnlPtr_devi,
          &(yRowId_devi[ptr]), &m,
          handle);
    HANDLE_ERROR(hipDeviceSynchronize());
    memcpyD2H<MemArrSizeType>(nnz_host, nnz_devi, 1);
    
    /* Backproject measurement on grid */
    CSRmv<val_t>()(handle, HIPSPARSE_OPERATION_TRANSPOSE,
          m, VGRIDSIZE, *nnz_host, &alpha, A, aVal_devi, aEcsrCnlPtr_devi, aVxlId_devi,
          &(yVal_devi[ptr]), &beta, x_devi);
    HANDLE_ERROR(hipDeviceSynchronize());
  }
#if MEASURE_TIME
  clock_t time3 = clock();
  printTimeDiff(time3, time2, "Time for BP: ");
#endif /* MEASURE_TIME */
  
  /* Normalize */
  val_t norm = sum<val_t>(x_devi, VGRIDSIZE);
  std::cout << "Norm: " << norm << std::endl;
  HANDLE_ERROR(hipDeviceSynchronize());
  scales<val_t>(x_devi, val_t(1./norm), VGRIDSIZE);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  /* Copy back to host */
  memcpyD2H<val_t>(&x_host[0], x_devi, VGRIDSIZE);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  /* Write to file */
  writeHDF5_Density(&x_host[0], on, grid);
  
  /* Cleanup */
  hipFree(yRowId_devi);
  hipFree(yVal_devi);
  hipsparseDestroy(handle);
  hipsparseDestroyMatDescr(A);
  hipFree(x_devi);
  hipFree(aCnlId_devi);
  hipFree(aCsrCnlPtr_devi);
  hipFree(aEcsrCnlPtr_devi);
  hipFree(aVxlId_devi);
  hipFree(aVal_devi);
  hipFree(nnz_devi);
  
#if MEASURE_TIME
  clock_t time4 = clock();
  printTimeDiff(time4, time3, "Time after BP: ");
#endif /* MEASURE_TIME */
  
  return 0;
}

