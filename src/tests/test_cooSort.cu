/**
 * Copyright 2016 Malte Zacharias
 *
 * This file is part of raptr.
 *
 * raptr is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * raptr is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with raptr.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/**
 * @file test_cooSort.cu
 */
/* 
 * Author: malte
 *
 * Created on 3. Februar 2015, 10:08
 */

#include <iostream>

#include "CUDA_HandleError.hpp"
#include "cooSort.hpp"

int main(int argc, char** argv) {
  // Create host arrays
  int A_host[6] = {1, 3, 2, 2, 1, 3};
  int B_host[6] = {2, 2, 1, 2, 1, 1};
  int C_host[6] = {2, 6, 3, 4, 1, 5};
  
  // Create and copy into device arrays
  int * A_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&A_devi, sizeof(A_devi[0]) * 6));
  HANDLE_ERROR(hipMemcpy(A_devi, A_host, sizeof(A_devi[0]) * 6, hipMemcpyHostToDevice));
  int * B_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&B_devi, sizeof(B_devi[0]) * 6));
  HANDLE_ERROR(hipMemcpy(B_devi, B_host, sizeof(B_devi[0]) * 6, hipMemcpyHostToDevice));
  int * C_devi = NULL;
  HANDLE_ERROR(hipMalloc((void**)&C_devi, sizeof(C_devi[0]) * 6));
  HANDLE_ERROR(hipMemcpy(C_devi, C_host, sizeof(C_devi[0]) * 6, hipMemcpyHostToDevice));
  
  // Sort
  cooSort(C_devi, A_devi, B_devi, 6);  
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Copy back to host
  HANDLE_ERROR(hipMemcpy(A_host, A_devi, sizeof(A_devi[0]) * 6, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(B_host, B_devi, sizeof(B_devi[0]) * 6, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(C_host, C_devi, sizeof(C_devi[0]) * 6, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipDeviceSynchronize());
  
  // Print results
  for(int i=0; i<6; i++) {
    std::cout << "i: " << i
              << " A: " << A_host[i]
              << " B: " << B_host[i]
              << " C: " << C_host[i] << std::endl;
  }
  
  // Release memory
  HANDLE_ERROR(hipFree(A_devi));
  HANDLE_ERROR(hipFree(B_devi));
  HANDLE_ERROR(hipFree(C_devi));
  
  return 0;
}

