#include "hip/hip_runtime.h"
/**
 * Copyright 2016 Malte Zacharias
 *
 * This file is part of raptr.
 *
 * raptr is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * raptr is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with raptr.
 * If not, see <http://www.gnu.org/licenses/>.
 */

/** @file test_mlemOperations.cu */
/* Author: malte
 *
 * Created on 4. Februar 2015, 14:15 */

#include <iostream>
#include "CUDA_HandleError.hpp"
#include "mlemOperations.hpp"

#define N 10

template<typename T>
hipError_t malloc_devi(T * & devi, int const n) {
  return hipMalloc((void**)&devi, sizeof(devi[0]) * n);
}

template<typename T>
hipError_t memcpy_h2d(T * const devi, T const * const host, int const n) {
  return hipMemcpy(devi, host, sizeof(devi[0]) * n, hipMemcpyHostToDevice);
}

template<typename T>
hipError_t memcpy_d2h(T * const host, T const * const devi, int const n) {
  return hipMemcpy(host, devi, sizeof(devi[0]) * n, hipMemcpyDeviceToHost);
}

template<typename T>
hipError_t memcpy_d2d(T * const devi0, T const * const devi1, int const n) {
  return hipMemcpy(devi0, devi1, sizeof(devi0[0]) * n, hipMemcpyDeviceToDevice);
}

typedef float val_t;

int main(int argc, char** argv) {
  /* Create host arrays */
  val_t A_host[N]; val_t B_host[N]; val_t C_host[N]; val_t D_host[N];
  
  /* Fill host arrays */
  for(int i=0; i<N; i++) {
    A_host[i] = (i+1)*(i+1);
    C_host[i] = i+1;
    B_host[i] = 10;
    D_host[i] = 0;
  }
  
  /* Create device arrays */
  val_t * A_devi = NULL;
  HANDLE_ERROR(malloc_devi(A_devi, N));
  val_t * B_devi = NULL;
  HANDLE_ERROR(malloc_devi(B_devi, N));
  val_t * C_devi = NULL;
  HANDLE_ERROR(malloc_devi(C_devi, N));
  val_t * D_devi = NULL;
  HANDLE_ERROR(malloc_devi(D_devi, N));
  
  /* Copy to device arrays */
  HANDLE_ERROR(memcpy_h2d(A_devi, A_host, N));
  HANDLE_ERROR(memcpy_h2d(B_devi, B_host, N));
  HANDLE_ERROR(memcpy_h2d(C_devi, C_host, N));
  HANDLE_ERROR(memcpy_h2d(D_devi, D_host, N));
  
  /* Divides */
  divides<val_t>(D_devi, A_devi, C_devi, N);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  HANDLE_ERROR(memcpy_d2h(D_host, D_devi, N));
  HANDLE_ERROR(hipDeviceSynchronize());
  std::cout << "D = A / C = " << std::endl;
  for(int i=0; i<N; i++) { std::cout << D_host[i] << std::endl; }
  std::cout << std::endl;
  
  /* Divides multiplies */
  dividesMultiplies<val_t>(D_devi, A_devi, B_devi, C_devi, N);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  HANDLE_ERROR(memcpy_d2h(D_host, D_devi, N));
  HANDLE_ERROR(hipDeviceSynchronize());
  std::cout << "D = A / B * C = " << std::endl;
  for(int i=0; i<N; i++) { std::cout << D_host[i] << std::endl; }
  std::cout << std::endl;

  /* Sum */
  val_t norm = sum<val_t>(D_devi, N);
  HANDLE_ERROR(hipDeviceSynchronize());
  
  std::cout << "norm =" << std::endl << norm << std::endl << std::endl;
    
  /* Scales */
  scales<val_t>(D_devi, (1./norm), N);
  
  HANDLE_ERROR(memcpy_d2h(D_host, D_devi, N));
  HANDLE_ERROR(hipDeviceSynchronize());
  std::cout << "D = D * " << 1./norm << " = " << std::endl;
  for(int i=0; i<N; i++) { std::cout << D_host[i] << std::endl; }
  std::cout << std::endl;
  
  hipFree(A_devi);
  hipFree(B_devi);
  hipFree(C_devi);
  hipFree(D_devi);
  
  return 0;
}

