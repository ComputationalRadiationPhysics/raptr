#include "hip/hip_runtime.h"
/** @file reco.cu
 * 
 *  @brief Main file that implements the reconstruction by the iterative MLEM
 *  algorithm.
 */

#define NBLOCKS 32

#include "cuda_wrappers.hpp"
#include "wrappers.hpp"
#include "CUDA_HandleError.hpp"
#include "CUSPARSE_HandleError.hpp"
#include "measure_time.hpp"
#include "typedefs.hpp"
#include "csrmv.hpp"
#include "mlemOperations.hpp"
#include "RayGenerators.hpp"

#include <hipsparse.h>
#include <sstream>
#include <cstdlib>
#include <fstream>
#include <mpi.h>

/* [512 * 1024 * 1024 / 4] (512 MiB of float or int); max # of elems in COO
 * matrix arrays on GPU */
MemArrSizeType const LIMBYTES(512*1024*1024);
MemArrSizeType const LIMNNZ(LIMBYTES/MemArrSizeType(sizeof(val_t)));

/* Max # of channels in COO matrix arrays */
ListSizeType const LIMM(LIMNNZ/VGRIDSIZE);

int main(int argc, char** argv) {
  
#if MEASURE_TIME
  clock_t time1 = clock();
#endif /* MEASURE_TIME */

  int mpi_rank;
  int mpi_size;
  MPI_Init(&argc, &argv);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  MPI_Comm_size(MPI_COMM_WORLD, &mpi_size);
  
  int const nargs(6);
  if(argc!=nargs+1) {
    std::cerr << "Error: Wrong number of arguments. Exspected: "
              << nargs << ":" << std::endl
              << "  filename of measurement" << std::endl
              << "  filename of output" << std::endl
              << "  number of rays" << std::endl
              << "  filename of sensitivity" << std::endl
              << "  number reco iterations" << std::endl
              << "  filename of density guess" << std::endl;
    exit(EXIT_FAILURE);
  }
  std::string const fn(argv[1]);
  std::string const on(argv[2]);
  std::string const sfn(argv[4]);
  int const nIt(atoi(argv[5]));
  std::string const xfn(argv[6]);
  
  /* NUMBER OF RAYS PER CHANNEL */
  int const nrays(atoi(argv[3]));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nrays_const), &nrays, sizeof(int)));
  
  /* MEASUREMENT SETUP */
  MS setup = MS(POS0X, POS1X, NA, N0Z, N0Y, N1Z, N1Y, DA, SEGX, SEGY, SEGZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(setup_const), &setup, sizeof(MS)));
  
  /* VOXEL GRID */
  VG grid = VG(GRIDOX, GRIDOY, GRIDOZ, GRIDDX, GRIDDY, GRIDDZ, GRIDNX, GRIDNY, GRIDNZ);
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(grid_const), &grid, sizeof(grid)));
  
  
  
  /* MEASUREMENT VECTOR Y */
  /* Number of non-zeros, row indices, values. */
  ListSizeType effM; std::vector<int> yRowId_host; std::vector<val_t> yVal_host;
  
  {
    int tmp_effM(0);
    readHDF5_MeasVct(yRowId_host, yVal_host, tmp_effM, fn);
    effM = ListSizeType(tmp_effM);
  }

  int * yRowId_devi = NULL;
  val_t * yVal_devi = NULL;
  mallocD_SparseVct(yRowId_devi, yVal_devi, effM);
  cpyH2DAsync_SparseVct(yRowId_devi, yVal_devi, &yRowId_host[0], &yVal_host[0], effM);
  
  /* SIMULATED MEASUREMENT VECTOR */
  val_t * yTildeVal_devi = NULL;
  mallocD(yTildeVal_devi, LIMM);
  
  /* "ERROR" */
  val_t * eVal_devi = NULL; 
  mallocD(eVal_devi, LIMM);
  
  
  
  /* STUFF FOR MV */
  hipsparseHandle_t handle = NULL; hipsparseMatDescr_t A = NULL;
  HANDLE_CUSPARSE_ERROR(hipsparseCreate(&handle));
  HANDLE_CUSPARSE_ERROR(hipsparseCreateMatDescr(&A));
  HANDLE_CUSPARSE_ERROR(customizeMatDescr(A, handle));
  val_t zero = val_t(0.); val_t one = val_t(1.);
  
  /* MAX NUMBER OF NON_ZEROS IN SYSTEM MATRIX */
  MemArrSizeType maxNnz(MemArrSizeType(effM) * MemArrSizeType(VGRIDSIZE));
  
  
  
  /* DENSITY X */
  bool xfile_good(false);
  {
    std::ifstream xfile(xfn.c_str());
    xfile_good = xfile.is_open();
  }
  
  std::vector<val_t> x_host;
  if(xfile_good) {
    std::cout << "Will use density from file " << xfn << std::endl;
    x_host = readHDF5_Density<val_t>(xfn);
  } else {
    std::cout << "No valid density input file given. Will use homogenous density." << std::endl;
    x_host = std::vector<val_t>(VGRIDSIZE, 1.);
  }
  
  val_t * x_devi = NULL;
  mallocD<val_t>(x_devi, VGRIDSIZE);
  memcpyH2D<val_t>(x_devi, &x_host[0], VGRIDSIZE);
  if(!xfile_good) {
    val_t norm = sum<val_t>(x_devi, VGRIDSIZE);
    HANDLE_ERROR(hipDeviceSynchronize());
    scales<val_t>(x_devi, (1./norm), VGRIDSIZE);
    HANDLE_ERROR(hipDeviceSynchronize());
  }
  
  /* INTERMEDIATE DENSITY GUESS */
  val_t * xx_devi = NULL;
  mallocD(xx_devi, VGRIDSIZE);
  
  /* CORRECTION */
  std::vector<val_t> c_host(VGRIDSIZE, 0.);
  std::vector<val_t> cMpi(  VGRIDSIZE, 0.);
  val_t * c_devi = NULL;
  mallocD(c_devi, VGRIDSIZE);
  
  /* SENSITIVITY */
  std::vector<val_t> s_host;
  int read_is_good(1);
  int sSize;
  std::cout << "Read from file " << sfn << std::endl;
  if(mpi_rank == 0) {
    s_host = readHDF5_Density<val_t>(sfn);
    if(s_host.size() == VGRIDSIZE) {
      read_is_good = 0;
    }
    sSize = s_host.size();
    {
      val_t S(0.); for(int i=0; i<VGRIDSIZE; i++) S+=s_host[i];
      std::cout << "S: " << S << std::endl;
    }
  }
  MPI_Bcast(&read_is_good, 1, MPI_INT, 0, MPI_COMM_WORLD);
  if(read_is_good != 0) {
    MPI_Finalize();
    std::cerr << "Error: Something about the sensitivity file (size?) is wrong"
              << std::endl;
    exit(EXIT_FAILURE);
  }
  MPI_Bcast(&sSize, 1, MPI_INT, 0, MPI_COMM_WORLD);
  if(mpi_rank != 0) {
    s_host.resize(sSize);
  }
  MPI_Bcast(&s_host[0], VGRIDSIZE, MPI_FLOAT, 0, MPI_COMM_WORLD);
  val_t * s_devi = NULL;
  mallocD(s_devi, VGRIDSIZE);
  memcpyH2DAsync<val_t>(s_devi, &s_host[0], VGRIDSIZE);
  
  /* Normalize */
  val_t norm = sum<val_t>(s_devi, VGRIDSIZE);
  scales<val_t>(s_devi, val_t(1./norm), VGRIDSIZE);
  
  
  
  /* SYSTEM MATRIX */
  /* Row (channel) ids, row pointers, effective row pointers, column (voxel)
   * ids, values, number of non-zeros (host, devi) */
  int * aCnlId_devi = NULL; int * aCsrCnlPtr_devi = NULL;
  int * aEcsrCnlPtr_devi = NULL; int * aVxlId_devi = NULL;
  val_t * aVal_devi = NULL;
  mallocD_SystemMatrix<val_t>(aCnlId_devi, aCsrCnlPtr_devi,
        aEcsrCnlPtr_devi, aVxlId_devi, aVal_devi, NCHANNELS, LIMM, VGRIDSIZE);
  MemArrSizeType * nnz_devi = NULL;
  mallocD<MemArrSizeType>(nnz_devi,          1);
  
#if MEASURE_TIME
  clock_t * itTimes = new clock_t[nIt+1];
  itTimes[0] = clock();
  if(mpi_rank == 0)
    printTimeDiff(itTimes[0], time1, "Time before reco iterations: ");
#endif /* MEASURE_TIME */

  /* How many chunks are needed? */
  ChunkGridSizeType NChunks(nChunks<ChunkGridSizeType, MemArrSizeType>(maxNnz, MemArrSizeType(LIMM*VGRIDSIZE)));
  
  /* RECO ITERATIONS */
  for(int it=0; it<nIt; it++) {
    
    /* Correction to zero */
    for(int i=0; i<VGRIDSIZE; i++) { c_host[i]=0; };
    memcpyH2D<val_t>(c_devi, &c_host[0], VGRIDSIZE);
    
    /* CHUNKWISE */
    ChunkGridSizeType chunkId = ChunkGridSizeType(mpi_rank);
    while(chunkId < NChunks) {
      ListSizeType m   = nInChunk(chunkId, effM, LIMM);
      ListSizeType ptr = chunkPtr(chunkId, LIMM);

      MemArrSizeType nnz_host[1] = {0};
      memcpyH2DAsync<MemArrSizeType>(nnz_devi, nnz_host, 1);

      /* Get system matrix */
      systemMatrixCalculation<val_t, ListSizeType, int, MemArrSizeType,
              RandRayGen<val_t, Trafo0_inplace, Trafo1_inplace> > (
            aEcsrCnlPtr_devi, aVxlId_devi, aVal_devi,
            nnz_devi,
            aCnlId_devi, aCsrCnlPtr_devi,
            &(yRowId_devi[ptr]), &m,
            handle);
      HANDLE_ERROR(hipDeviceSynchronize());
      memcpyD2H<MemArrSizeType>(nnz_host, nnz_devi, 1);

      /* Simulate measurement */
      CSRmv<val_t>()(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            m, VGRIDSIZE, *nnz_host, &one, A, aVal_devi, aEcsrCnlPtr_devi, aVxlId_devi,
            x_devi, &zero, yTildeVal_devi);
      HANDLE_ERROR(hipDeviceSynchronize());

      /* Calculate "error" */
      divides<val_t>(eVal_devi, &(yVal_devi[ptr]), yTildeVal_devi,
            m);
      HANDLE_ERROR(hipDeviceSynchronize());

      /* Backproject error */
      CSRmv<val_t>()(handle, HIPSPARSE_OPERATION_TRANSPOSE,
            m, VGRIDSIZE, *nnz_host, &one, A, aVal_devi, aEcsrCnlPtr_devi, aVxlId_devi,
            eVal_devi, &one, c_devi);
      HANDLE_ERROR(hipDeviceSynchronize());
      
      /* Go for next chunk */
      chunkId += mpi_size;
    
    } /* while(chunkId < NChunks) */
    
    /* Reduce correction between processes */
    memcpyD2H(&c_host[0], c_devi, VGRIDSIZE);
    MPI_Allreduce(&c_host[0], &cMpi[0], VGRIDSIZE, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);
    memcpyH2D(c_devi, &cMpi[0], VGRIDSIZE);
    
    /* Print stuff */
    if(mpi_rank == 0) {
      
      std::cout << "Sum of c_devi: " << sum<val_t>(c_devi, VGRIDSIZE) << std::endl << std::flush;
      std::cout << "Sum of s_devi: " << sum<val_t>(s_devi, VGRIDSIZE) << std::endl << std::flush;
      std::cout << "Sum of x_devi: " << sum<val_t>(x_devi, VGRIDSIZE) << std::endl << std::flush;
      std::cout << "Sum of xx_devi: " << sum<val_t>(xx_devi, VGRIDSIZE) << std::endl << std::flush;
    
    } /* if(mpi_rank == 0) */
    
    /* Improve guess */
    dividesMultiplies<val_t>(xx_devi, x_devi, c_devi, s_devi, VGRIDSIZE);
    HANDLE_ERROR(hipDeviceSynchronize());
    
    /* Copy */
    memcpyD2D(x_devi, xx_devi, VGRIDSIZE);

    /* Normalize */
    val_t norm = sum<val_t>(x_devi, VGRIDSIZE);
    scales<val_t>(x_devi, val_t(1./norm), VGRIDSIZE);
    std::cout << "Norm: " << sum<val_t>(x_devi, VGRIDSIZE) << std::endl << std::flush;
  
    /* Write to file */
    if(mpi_rank==0) {
      
      memcpyD2H<val_t>(&x_host[0], x_devi, VGRIDSIZE);
      std::stringstream ss("");
      ss << it;
      writeHDF5_Density(&x_host[0], ss.str() + std::string("_") + on, grid);
      
#if MEASURE_TIME
      itTimes[it+1] = clock();
      if(mpi_rank == 0)
        printTimeDiff(itTimes[it+1], itTimes[it], "Time for latest reco iteration: ");
#endif
      
    } /* if(mpi_rank == 0) */
     
  } /* for(int it=0; it<nIt; it++) */

#if MEASURE_TIME
  clock_t time3 = clock();
  if(mpi_rank == 0)
    printTimeDiff(time3, itTimes[0], "Time for reco iterations: ");
  delete[] itTimes;
#endif /* MEASURE_TIME */
    
  /* Cleanup */
  hipFree(yRowId_devi);
  hipFree(yVal_devi);
  hipsparseDestroy(handle);
  hipsparseDestroyMatDescr(A);
  hipFree(x_devi);
  hipFree(aCnlId_devi);
  hipFree(aCsrCnlPtr_devi);
  hipFree(aEcsrCnlPtr_devi);
  hipFree(aVxlId_devi);
  hipFree(aVal_devi);
  hipFree(nnz_devi);
    
  MPI_Finalize();
  
#if MEASURE_TIME
  clock_t time4 = clock();
  if(mpi_rank == 0)
    printTimeDiff(time4, time3, "Time after reco iterations: ");
#endif /* MEASURE_TIME */
  
  return 0;
}

